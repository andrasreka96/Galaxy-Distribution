
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>


//histogram size
size_t HIST_SIZE = 360 * sizeof(unsigned long long int);
//galaxy number
long int N = 100000;
//input data
int    NoofReal;
int    NoofRand;
float *real_rasc, *real_decl;
float *rand_rasc, *rand_decl;
long int CPUMemory = 0L;
long int GPUMemory = 0L;
int  readdata(char *, char *);


__global__  void calcHist(unsigned long long int *DR,unsigned long long int *DD, unsigned long long int *RR, float *real_rasc, float *real_decl, float *rand_rasc, float *rand_decl, long int N)
{
   long int blockId = blockIdx.x
      + blockIdx.y * gridDim.x
      + gridDim.x * gridDim.y * blockIdx.z;

   long int threadId = blockId*(blockDim.x*blockDim.y*blockDim.z)
      + (threadIdx.z *(blockDim.x * blockDim.y))
      + (threadIdx.y * blockDim.x)
      + threadIdx.x;

   // printf("Thread id: %d\n", threadId);

   if (threadId >= 2*N*N) return;

   //calculate DR
   if (threadId < N * N){
      // printf("Calculate DR (%d, %d) pair from total %d\n", threadId/N, threadId%N, N);

      float acos_in = sin(real_decl[threadId/N])*sin(rand_decl[threadId%N])+cos(real_decl[threadId/N])*cos(rand_decl[threadId%N])*cos(real_rasc[threadId/N]-rand_rasc[threadId%N]);
      if (acos_in >= 0)
         acos_in = fmin(acos_in,1);
      else
         acos_in = fmax(acos_in,-1);
      
      atomicAdd(&DR[(int)(acos(acos_in) * 180 / acosf(-1.0f)*4)], 1);
   }else{

      long int i = (threadId - N*N)/N;
      long int j = threadId % N;

      if (i == j) return;

      if (i < j){
         // compute DD

         // printf("Calculate DD (%d, %d) pair from total %d\n", i, j, N);

         float acos_in = sin(real_decl[i])*sin(real_decl[j])+cos(real_decl[i])*cos(real_decl[j])*cos(real_rasc[i]-real_rasc[j]);
         if (acos_in >= 0)
            acos_in = fmin(acos_in,1);
         else
            acos_in = fmax(acos_in,-1);
         
         atomicAdd(&DD[(int)(acos(acos_in)* 180 / acosf(-1.0f) * 4)], 2);

      }
      else{
         // compute RR

         // printf("Calculate RR (%d, %d) pair from total %d\n", j, i, N);

         float acos_in = sin(rand_decl[j])*sin(rand_decl[i])+cos(rand_decl[j])*cos(rand_decl[i])*cos(rand_rasc[j]-rand_rasc[i]);
         if (acos_in >= 0)
            acos_in = fmin(acos_in,1);
         else
            acos_in = fmax(acos_in,-1);
         
         atomicAdd(&RR[(int)(acos(acos_in)* 180 / acosf(-1.0f) *  4)], 2);

      }

   }

}
void calcHistWrep(unsigned long long int *DR,unsigned long long int *DD, unsigned long long int *RR)
{
   unsigned long long int *histogramDR_gpu, *histogramDD_gpu, *histogramRR_gpu;
   float *real_rasc_gpu, *real_decl_gpu;
   float *rand_rasc_gpu, *rand_decl_gpu;
   long int *n;
  
   // input data is available in the arrays float real_rasc[], real_decl[], rand_rasc[], rand_decl[];
   // allocate memory on the GPU for input data
   hipMalloc(&real_rasc_gpu, N * sizeof(float));
   hipMalloc(&real_decl_gpu, N * sizeof(float));
   hipMalloc(&rand_rasc_gpu, N * sizeof(float));
   hipMalloc(&rand_decl_gpu, N * sizeof(float));
   
   // the number of galaxies is needed because of the work distribution logic
   hipMalloc((void**)&n, sizeof(long int));

    // allocate memory on the GPU for histograms
   hipMalloc((void **)&histogramDR_gpu, HIST_SIZE);
   hipMalloc((void **)&histogramDD_gpu, HIST_SIZE);
   hipMalloc((void **)&histogramRR_gpu, HIST_SIZE);
   hipMemset(histogramDR_gpu, 0, HIST_SIZE);
   hipMemset(histogramDD_gpu, 0, HIST_SIZE);
   hipMemset(histogramRR_gpu, 0, HIST_SIZE);

   // initialize the data on GPU by copying the real and rand data to the GPU
   hipMemcpy(real_rasc_gpu, real_rasc, N * sizeof(float), hipMemcpyHostToDevice);
   hipMemcpy(real_decl_gpu, real_decl, N * sizeof(float), hipMemcpyHostToDevice);
   hipMemcpy(rand_rasc_gpu, rand_rasc, N * sizeof(float), hipMemcpyHostToDevice);
   hipMemcpy(rand_decl_gpu, rand_decl, N * sizeof(float), hipMemcpyHostToDevice);
   hipMemcpy(n, &N, sizeof(long int), hipMemcpyHostToDevice);
 
   long int threadsInBlock = 32*32;
   long int blocksInGrid = ( 2*N*N + threadsInBlock - 1 )/threadsInBlock;
   printf("Size of the blocks in grid: %ld\nThread number in each block:%ld\n",  blocksInGrid, threadsInBlock );
   calcHist<<<  blocksInGrid, threadsInBlock >>>(histogramDR_gpu,histogramDD_gpu,histogramRR_gpu, real_rasc_gpu, real_decl_gpu, rand_rasc_gpu, rand_decl_gpu, N);

   //move back the results from the gpu memory
   hipMemcpy(DR, histogramDR_gpu, HIST_SIZE, hipMemcpyDeviceToHost);
   hipMemcpy(DD, histogramDD_gpu, HIST_SIZE, hipMemcpyDeviceToHost);
   hipMemcpy(RR, histogramRR_gpu, HIST_SIZE, hipMemcpyDeviceToHost);
  
}

int main(int argc, char** argv){

   struct timeval _ttime;
   struct timezone _tzone;

   gettimeofday(&_ttime, &_tzone);
   double time_start = (double)_ttime.tv_sec + (double)_ttime.tv_usec/1000000.;

   //read real and random galaxies
   if ( readdata(argv[1], argv[2]) != 0 ) return(-1);

   // init global histogram arrays
   unsigned long long int *histogramDR, *histogramDD, *histogramRR;

   //allocate memory for the global histograms
   histogramDR=(unsigned long long int *) malloc(HIST_SIZE);
   histogramDD=(unsigned long long int *) malloc(HIST_SIZE);
   histogramRR=(unsigned long long int *) malloc(HIST_SIZE);

   //run the GPU kernel
   calcHistWrep(histogramDR, histogramDD, histogramRR);

   // galaxie distance of itself will be 0    
   histogramDD[0] += N;
   histogramRR[0] += N;    

    
   // check point: the sum of all historgram entries should be galaxy_num**2 
   long int histsum = 0L;

    for ( int i = 0; i < 360; ++i ) histsum += histogramDD[i];
    printf("   Histogram DD : sum = %ld\n",histsum);
    if ( histsum != (N*N)) {printf("   Histogram sums should be %ld. Ending program prematurely\n", N*N);return(0);}

    histsum = 0L;
    for ( int i = 0; i < 360; ++i ) histsum += histogramRR[i];
    printf("   Histogram RR : sum = %ld\n",histsum);
    if ( histsum != (N*N)) {printf("   Histogram sums should be %ld. Ending program prematurely\n", N*N);return(0);}

    histsum = 0L;
    for ( int i = 0; i < 360; ++i ) histsum += histogramDR[i];
    printf("   Histogram DR : sum = %ld\n",histsum);
    if ( histsum != N*N ) {printf("   Histogram sums should be %ld. Ending program prematurely\n", N*N);return(0);}

    printf("   Omega values for the histograms:\n");
    float omega[360];
    for ( int i = 0; i < 10; ++i ) 
        if ( (long long int)histogramRR[i] != 0LL )
           {
           omega[i] = ((long long int)histogramDD[i] - 2L*(long long int)histogramDR[i] + (long long int)histogramRR[i])/((float)(histogramRR[i]));
           if ( i < 10 ) printf("      angle %.2f deg. -> %.2f deg. : %.3f\n", i*0.25, (i+1)*0.25, omega[i]);
           }

    FILE *out_file = fopen(argv[3],"w");
    if ( out_file == NULL ) printf("   ERROR: Cannot open output file %s\n",argv[3]);
    else
       {
       for ( int i = 0; i < 360; ++i ) 
           if ( (long long int)histogramRR[i] != 0LL )
              fprintf(out_file,"%.2f  : %.3f\n", i*0.25, omega[i] ); 
       fclose(out_file);
       printf("   Omega values written to file %s\n",argv[3]);
       }

    free(real_rasc); free(real_decl);
    free(rand_rasc); free(rand_decl);

    gettimeofday(&_ttime, &_tzone);
    double time_end = (double)_ttime.tv_sec + (double)_ttime.tv_usec/1000000.;

    printf("   Wall clock run time    = %.1lf secs\n",time_end - time_start);


   return 0;

}

int readdata(char *argv1, char *argv2)
{
  int    i,linecount;
  char   inbuf[80];
  double ra, dec, dpi;
  FILE  *infil;
                                         
  printf("   Assuming data is in arc minutes!\n");
                          // phi   = ra/60.0 * dpi/180.0;
                          // theta = (90.0-dec/60.0)*dpi/180.0;
                          // otherwise use 
                          // phi   = ra * dpi/180.0;
                          // theta = (90.0-dec)*dpi/180.0;

  dpi = acos(-1.0);
  infil = fopen(argv1,"r");
  if ( infil == NULL ) {printf("Cannot open input file %s\n",argv1);return(-1);}

  linecount =0;
  while ( fgets(inbuf,80,infil) != NULL ) ++linecount;
  rewind(infil);

  printf("   %s contains %d galaxies\n",argv1, linecount-1);

  NoofReal = linecount-1;

  if ( NoofReal != 100000 ) {printf("Incorrect number of galaxies\n");return(1);}

  real_rasc = (float *)calloc(NoofReal,sizeof(float));
  real_decl = (float *)calloc(NoofReal,sizeof(float));
  CPUMemory += 2L*NoofReal*sizeof(float);

  fgets(inbuf,80,infil);
  sscanf(inbuf,"%d",&linecount);
  if ( linecount != 100000 ) {printf("Incorrect number of galaxies\n");return(1);}

  i = 0;
  while ( fgets(inbuf,80,infil) != NULL )
      {
      if ( sscanf(inbuf,"%lf %lf",&ra,&dec) != 2 ) 
         {
         printf("   Cannot read line %d in %s\n",i+1,argv1);
         fclose(infil);
         return(-1);
         }
      real_rasc[i] = (float)( ra/60.0*dpi/180.0);
      real_decl[i] = (float)(dec/60.0*dpi/180.0);
      ++i;
      }

  fclose(infil);

  if ( i != NoofReal ) 
      {
      printf("   Cannot read %s correctly\n",argv1);
      return(-1);
      }

  infil = fopen(argv2,"r");
  if ( infil == NULL ) {printf("Cannot open input file %s\n",argv2);return(-1);}

  linecount =0;
  while ( fgets(inbuf,80,infil) != NULL ) ++linecount;
  rewind(infil);

  printf("   %s contains %d galaxies\n",argv2, linecount-1);

  NoofRand = linecount-1;
  if ( NoofRand != 100000 ) {printf("Incorrect number of random galaxies\n");return(1);}

  rand_rasc = (float *)calloc(NoofRand,sizeof(float));
  rand_decl = (float *)calloc(NoofRand,sizeof(float));
  CPUMemory += 2L*NoofRand*sizeof(float);

  fgets(inbuf,80,infil);
  sscanf(inbuf,"%d",&linecount);
  if ( linecount != 100000 ) {printf("Incorrect number of random galaxies\n");return(1);}

  i =0;
  while ( fgets(inbuf,80,infil) != NULL )
      {
      if ( sscanf(inbuf,"%lf %lf",&ra,&dec) != 2 ) 
         {
         printf("   Cannot read line %d in %s\n",i+1,argv2);
         fclose(infil);
         return(-1);
         }
      rand_rasc[i] = (float)( ra/60.0*dpi/180.0);
      rand_decl[i] = (float)(dec/60.0*dpi/180.0);
      ++i;
      }

  fclose(infil);

  if ( i != NoofReal ) 
      {
      printf("   Cannot read %s correctly\n",argv2);
      return(-1);
      }

  return(0);
}